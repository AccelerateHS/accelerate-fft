/*
 * Module      : Twine
 * Copyright   : [2016] Trevor L. McDonell
 * License     : BSD3
 *
 * Maintainer  : Trevor L. McDonell <tmcdonell@cse.unsw.edu.au>
 * Stability   : experimental
 * Portability : non-portable (GHC extensions)
 *
 * Convert between Accelerate's Struct-of-Array representation of complex
 * numbers and the Array-of-Struct representation necessary for CUBLAS.
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#ifdef __cplusplus
extern "C" {
#endif

__global__ void interleave
(
    hipFloatComplex * __restrict__ cplx,
    const float * __restrict__ real,
    const float * __restrict__ imag,
    const int size
)
{
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < size; ix += gridSize) {
      const float re = real[ix];
      const float im = imag[ix];

      cplx[ix] = make_hipFloatComplex(re, im);
    }
}

__global__ void deinterleave
(
    float * __restrict__ real,
    float * __restrict__ imag,
    const hipFloatComplex * __restrict__ cplx,
    const int size
)
{
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < size; ix += gridSize) {
      const hipFloatComplex c = cplx[ix];

      real[ix] = hipCrealf(c);
      imag[ix] = hipCimagf(c);
    }
}

#ifdef __cplusplus
}
#endif

