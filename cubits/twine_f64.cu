/*
 * Module      : Twine
 * Copyright   : [2016] Trevor L. McDonell
 * License     : BSD3
 *
 * Maintainer  : Trevor L. McDonell <tmcdonell@cse.unsw.edu.au>
 * Stability   : experimental
 * Portability : non-portable (GHC extensions)
 *
 * Convert between Accelerate's Struct-of-Array representation of complex
 * numbers and the Array-of-Struct representation necessary for CUBLAS.
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#ifdef __cplusplus
extern "C" {
#endif

__global__ void interleave
(
    hipDoubleComplex * __restrict__ cplx,
    const double * __restrict__ real,
    const double * __restrict__ imag,
    const int size
)
{
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < size; ix += gridSize) {
      const double re = real[ix];
      const double im = imag[ix];

      cplx[ix] = make_hipDoubleComplex(re, im);
    }
}

__global__ void deinterleave
(
    double * __restrict__ real,
    double * __restrict__ imag,
    const hipDoubleComplex * __restrict__ cplx,
    const int size
)
{
    const int gridSize = blockDim.x * gridDim.x;
    int ix;

    for (ix = blockDim.x * blockIdx.x + threadIdx.x; ix < size; ix += gridSize) {
      const hipDoubleComplex c = cplx[ix];

      real[ix] = hipCreal(c);
      imag[ix] = hipCimag(c);
    }
}

#ifdef __cplusplus
}
#endif

